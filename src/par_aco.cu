#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "par_aco.cuh"

// Print the iter_t structure
__host__ __device__ void print_iter(iter_t iter, int num_nodes);

__device__ int sample(int k, int *ints, float *weights);

// Compute the edge attractiveness matrix given the graph, tau, eta, a, and b.
// Store the output in `float *A`
__device__ void edge_attractiveness(float *A, float *adjacency_matrix, int num_nodes,
        float *tau, float *eta, float a, float b);

// Run a single ant, which will update tau and return an iter_t
__device__ iter_t run_ant(float *adjacency_matrix, int num_nodes, float *tau, float *A,
        iter_t iter);

// Run the ant colony optimization algorithm on a graph.
__global__ void tour_construction(float *adjacency_matrix, float *A, int num_nodes,
        float a, float b, float p);

__global__ void pheromone_update(float *adjacency_matrix, float *tau, int num_nodes, int *tours, float a, float b, float p, int m);

